#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define KERNEL_SIZE 27
__constant__ unsigned char WEIGHT_BYTES[KERNEL_SIZE * KERNEL_SIZE * sizeof(double)];

#define NUM_THREADS_FORWARD 512
#define NUM_THREADS_BACKWARD 256

namespace {
template <typename scalar_t>
__global__ void sm_linear_3d_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> image, // 1, N, N, N
    torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> y,
    const int nblocks,
    const int N1, const int N2, const int N3) {

  __shared__ scalar_t z[NUM_THREADS_FORWARD];

  const scalar_t *WEIGHT = (const scalar_t *)(WEIGHT_BYTES);

  const int b = blockIdx.x / nblocks;
  const int innerBlock = blockIdx.x % nblocks;
  int location = blockDim.x * innerBlock + threadIdx.x;

  z[threadIdx.x] = 0.0;

  if (location < N1*N2*N3) {
    const int ij = location % N3;
    location /= N3;
    const int j = location % N2;
    const int i = location / N2;

    for (int k = 0; k <= 2; ++k) {
      for (int l = 0; l <= 2; ++l) {
        for (int kl = 0; kl <= 2; ++kl) {
          z[threadIdx.x] += WEIGHT[27*b+9*k+3*l+kl] * image[0][i+k][j+l][ij+kl];
        }
      }
    }
  }
  __syncthreads();

  // reduction
  int data = blockDim.x;
  while (data > 1) {
    if (threadIdx.x < data / 2)
      z[threadIdx.x] += z[threadIdx.x + data / 2];
    data /= 2;
    __syncthreads();
  }
  if (threadIdx.x == 0) atomicAdd(&y[0], z[0]);
}


template <typename scalar_t>
__global__ void sm_linear_3d_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> grad_output, // bs, 1, N, N, N
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> image, // 1, N, N, N
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_w,
    const int locationsPerBlock) {

  __shared__ scalar_t d_w[KERNEL_SIZE*KERNEL_SIZE];

  const int N1 = (image.size(1) - 2) / locationsPerBlock;
  const int N2 = (image.size(2) - 2) / locationsPerBlock;
  const int N3 = (image.size(3) - 2) / locationsPerBlock;

  const int nBlocksPerCopy = (KERNEL_SIZE*KERNEL_SIZE + blockDim.x - 1) / blockDim.x;

  const int block = blockIdx.x / nBlocksPerCopy;
  const int innerBlock = blockIdx.x % nBlocksPerCopy;

  int location = block;
  const int ij = (location % N3) * locationsPerBlock;
  location /= N3;
  const int j = (location % N2) * locationsPerBlock;
  const int i = (location / N2) * locationsPerBlock;

  const int p = innerBlock * blockDim.x + threadIdx.x;

  if (p >= KERNEL_SIZE*KERNEL_SIZE) return; // Wasted threads

  const int pp = p % KERNEL_SIZE;
  const int kl = pp % 3;
  const int l = (pp / 3) % 3;
  const int k = pp / 9;


  const int p0 = p / KERNEL_SIZE;

  const int c = 27*p0+9*k+3*l+kl;
  d_w[c] = 0.0;

  for (int _i = 0; _i < locationsPerBlock; ++_i) {
    for (int _j = 0; _j < locationsPerBlock; ++_j) {
      for (int _ij = 0; _ij < locationsPerBlock; ++_ij) {
        d_w[c] += image[0][i+_i+k][j+_j+l][ij+_ij+kl];
      }
    }
  }

  atomicAdd(&grad_w[p0][0][k][l][kl], d_w[c]);
}

} // namespace

std::vector<torch::Tensor> sm_linear_3d_cuda_forward(
    torch::Tensor image,
    torch::Tensor weights,
    torch::Tensor bias) {

  if (image.dtype() == torch::ScalarType::Double) {
    hipMemcpyToSymbol(HIP_SYMBOL(WEIGHT_BYTES), weights.data_ptr<double>(), KERNEL_SIZE*KERNEL_SIZE * sizeof(double));
  } else {
    hipMemcpyToSymbol(HIP_SYMBOL(WEIGHT_BYTES), weights.data_ptr<float>(), KERNEL_SIZE*KERNEL_SIZE * sizeof(float));
  }

  const int N1 = image.size(1)-2;
  const int N2 = image.size(2)-2;
  const int N3 = image.size(3)-2;

  auto y = torch::zeros({1}, torch::dtype(image.dtype()).device(image.device()));

  const int nthreads = NUM_THREADS_FORWARD;
  const int nblocks = (N1*N2*N3 + nthreads - 1) / nthreads; // b, i, j, ij

  const dim3 threads(nthreads);
  const dim3 blocks(nblocks * KERNEL_SIZE);

  AT_DISPATCH_FLOATING_TYPES(image.type(), "sm_linear_3d_forward_cuda", ([&] {
    sm_linear_3d_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        image.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        y.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
        nblocks,
        N1, N2, N3);
  }));

  y /= KERNEL_SIZE * N1*N2*N3;
  y += bias.mean();
  return {y};
}

std::vector<torch::Tensor> sm_linear_3d_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor image) {

  const int N = image.size(2) - 2;
  const int totalData = N * N * N;

  const int nThreads = NUM_THREADS_BACKWARD;
  // const int nBlocksPerElement = (totalData + nThreads - 1) / nThreads; // number of blocks one element dW_{p, 0, m, n, mn} or db_{p}
  // const int nBlocks = nBlocksPerElement * (KERNEL_SIZE + 1) * KERNEL_SIZE; // dw size 27 * 27 and db size 27
  // const int nBlocks = (totalData + nThreads - 1) / nThreads;

  const int nBlocksPerCopy = (KERNEL_SIZE*KERNEL_SIZE + nThreads - 1) / nThreads;

  const int locationsPerBlock = 4;

  assert(N % locationsPerBlock == 0); // Data must be divisible by divisions

  const dim3 threads(nThreads);
  const dim3 blocks(nBlocksPerCopy*(totalData/std::pow(locationsPerBlock, 3)));

  auto grad_w = torch::zeros({27, 1, 3, 3, 3}, torch::dtype(image.dtype()).device(image.device()));
  auto grad_b = torch::ones({27}, torch::dtype(image.dtype()).device(image.device())) / KERNEL_SIZE * grad_output;

  AT_DISPATCH_FLOATING_TYPES(grad_output.type(), "sm_linear_3d_cuda_backward", ([&] {
    sm_linear_3d_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        grad_output.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
        image.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        grad_w.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        locationsPerBlock);
  }));
  grad_w /= KERNEL_SIZE * std::pow(N, 3);
  grad_w *= grad_output;
  return {grad_w, grad_b};
}
